#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <limits.h>
#include <algorithm>

#include <hip/hip_runtime.h>

using namespace std;

#define INF           INT_MAX-1

void init(float *matrix, int n)
{
    for (int i=0;i<n;i++) {
        for (int j=0;j<n;j++) {
            if(i==j)
            {
                matrix[i * n + j] = 0;
            }
            else
            {
                matrix[i * n + j] = INF;
            }
        }
    }
}

__global__
void FloydWarshall(int k, int i, float *matrix, int n)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x; /* This thread’s matrix column */

    if(col >= n)
        return;

    int arrayIndex = n * i + col; // Calculating D[i][j]

    float trkc = matrix[n * i + k];   // Calculating D[i][k]
    
    float tckr = matrix[k*n + col]; /* this column, kth row */
    
    matrix[arrayIndex] = fmin(matrix[arrayIndex], trkc + tckr);
}



int main(int argc, char *argv[])
{      
    char *arg_vertices = getenv("N_VERTICES");
    char *arg_threads_per_block = getenv("N_THREADS");
	
    size_t vertices = atoi(arg_vertices);
    int threads_per_block   = atoi(arg_threads_per_block);
   
    float *host_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
    
	init(host_matrix, vertices);
    
	for(int i = 0 ; i < vertices ; i++ ) {
		for(int j = 0 ; j< vertices; j++ ) {
            if( i == j )
                host_matrix[i * vertices + j] = 0;

            else {
				int num = i + j;

				if (num % 3 == 0)
					 host_matrix[i * vertices + j] = num / 2;
				else if (num % 2 == 0)
					 host_matrix[i * vertices + j] = num * 2;
				else
					 host_matrix[i * vertices + j] = num;
			}
		}
	}	

    
    size_t tot = vertices * vertices * sizeof(float);
    float *device_matrix = NULL;
    hipMalloc((float **)&device_matrix, tot);

    hipMemcpy(device_matrix, host_matrix, tot, hipMemcpyHostToDevice);

    int blocks_per_grid = vertices + (threads_per_block - 1) /threads_per_block;
    struct timeval tvalBefore, tvalAfter;
    gettimeofday (&tvalBefore, NULL);
    for(int via = 0; via < vertices; via++) {
	for(int j = 0; j < vertices; j++){

	    FloydWarshall<<<blocks_per_grid, threads_per_block>>>(via, j, device_matrix, vertices);
        hipDeviceSynchronize();
	}
    }
    gettimeofday (&tvalAfter, NULL);
    printf("Time: %ld microseconds\n",
        ((tvalAfter.tv_sec - tvalBefore.tv_sec)*1000000L
        +tvalAfter.tv_usec) - tvalBefore.tv_usec
        );
    float *result_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
 
    hipMemcpy(result_matrix, device_matrix, tot, hipMemcpyDeviceToHost);
    return 0;
}
