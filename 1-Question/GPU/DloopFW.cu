#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <limits.h>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

#define INF           INT_MAX-1

__global__
void FloydWarshall(int k, float *matrix, int n)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(col >= n)
        return;

    int index_i_j = n * blockIdx.y + col;

    __shared__ float matrix_value_i_k;

    if(threadIdx.x == 0)
        matrix_value_i_k = matrix[n * blockIdx.y + k];
    
    __syncthreads();
    
    float matrix_value_k_j = matrix[k*n + col];
    
    matrix[index_i_j] = fmin(matrix[index_i_j], matrix_value_i_k + matrix_value_k_j);
}



int main(int argc, char *argv[])
{      
    char *arg_vertices = getenv("N_VERTICES");
	
    size_t vertices = atoi(arg_vertices);
   
    float *host_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
    
	for(int i = 0 ; i < vertices ; i++ ) {
		for(int j = 0 ; j< vertices; j++ ) {
            if( i == j )
                host_matrix[i * vertices + j] = 0;

            else {
				int num = i + j;

				if (num % 3 == 0)
					 host_matrix[i * vertices + j] = num / 2;
				else if (num % 2 == 0)
					 host_matrix[i * vertices + j] = num * 2;
				else
					 host_matrix[i * vertices + j] = num;
			}
		}
	}	

    
    size_t tot = vertices * vertices * sizeof(float);
    float *device_matrix = NULL;
    hipMalloc((float **)&device_matrix, tot);

    hipMemcpy(device_matrix, host_matrix, tot, hipMemcpyHostToDevice);

    int threads_per_block   = 1024;
    dim3 blocks_per_grid((vertices + threads_per_block - 1) /
                                threads_per_block, vertices);
    struct timeval tvalBefore, tvalAfter;
    gettimeofday (&tvalBefore, NULL);
    for(int via = 0; via < vertices; via++) {

	    FloydWarshall<<<blocks_per_grid, threads_per_block>>>(via, device_matrix, vertices);
        hipDeviceSynchronize();
    }
    gettimeofday (&tvalAfter, NULL);

    printf("Time: %ld microseconds\n",
        ((tvalAfter.tv_sec - tvalBefore.tv_sec)*1000000L
        +tvalAfter.tv_usec) - tvalBefore.tv_usec
        );
    float *result_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
 
    hipMemcpy(result_matrix, device_matrix, tot, hipMemcpyDeviceToHost);
    return 0;
}
