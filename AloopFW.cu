#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <limits.h>
#include <algorithm>

#include <hip/hip_runtime.h>

using namespace std;

#define INF           INT_MAX-1

void init(float *matrix, int n)
{
    for (int i=0;i<n;i++) {
        for (int j=0;j<n;j++) {
            if(i==j)
            {
                matrix[i * n + j] = 0;
            }
            else
            {
                matrix[i * n + j] = INF;
            }
        }
    }
}

__global__
void FloydWarshall(float *matrix, int n)
{
        
	for(int via=0; via < n; via++) {
	    for(int from=0;from<n;from++) {
            for(int to=0;to<n;to++) {
                if(from!=to && from!=via && to!=via) {

				    matrix[from * n + to] = min(matrix[from * n + to], 
                            matrix[from * n + via] + matrix[via * n + to]);

			    }
                        
            }
        }
   	}
}


int main(int argc, char *argv[])
{      
    char *arg_vertices = getenv("N_VERTICES");
	
    size_t vertices = atoi(arg_vertices);
   
    float *host_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
    
	init(host_matrix, vertices);
    
	for(int i = 0 ; i < vertices ; i++ ) {
		for(int j = 0 ; j< vertices; j++ ) {
            if( i == j )
                host_matrix[i * vertices + j] = 0;

            else {
				int num = i + j;

				if (num % 3 == 0)
					 host_matrix[i * vertices + j] = num / 2;
				else if (num % 2 == 0)
					 host_matrix[i * vertices + j] = num * 2;
				else
					 host_matrix[i * vertices + j] = num;
			}
		}
	}	

    
    size_t tot = vertices * vertices * sizeof(float);
    float *device_matrix = NULL;
    hipMalloc((float **)&device_matrix, tot);

    hipMemcpy(device_matrix, host_matrix, tot, hipMemcpyHostToDevice);

	//FloydWarshall(host_matrix, vertices);
	FloydWarshall<<<1, 1>>>(device_matrix, vertices);

    float *result_matrix =(float *)malloc( vertices * vertices *
                sizeof(float));
 
    hipMemcpy(result_matrix, device_matrix, tot, hipMemcpyDeviceToHost);
    
    for(int i = 0 ; i < vertices; i++ ) 
	{
		cout << "\n";
		for(int j = 0 ; j< vertices ;j++ )
			cout << result_matrix[i * vertices + j] << " " ;
	} 

	return 0;
}
